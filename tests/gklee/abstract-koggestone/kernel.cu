#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>

#ifndef N
#error N must be defined
#endif

#if rwidth == 8
  #define rtype uchar2
#elif rwidth == 16
  #define rtype ushort2
#elif rwidth == 32
  #define rtype uint2
#elif rwidth == 64
  #define rtype ulong2
#else
  #error rwidth must be defined
#endif

#ifdef _SYM
  #define ASSERT_NO_CUDA_ERROR( callReturningErrorstatus ) callReturningErrorstatus
#else
  #define ASSERT_NO_CUDA_ERROR( callReturningErrorstatus ) {     \
    hipError_t err = callReturningErrorstatus;                  \
    if (err != hipSuccess) {                                    \
      fprintf(stderr,                                            \
              "Cuda error (%s/%d) in file '%s' in line %i\n",    \
              hipGetErrorString(err), err, __FILE__, __LINE__); \
      exit(1);                                                   \
    }                                                            \
  } while(0);
#endif

__global__ void koggestone(rtype *len, rtype *out, unsigned *error) {
  __shared__ rtype result[N];

  unsigned t = threadIdx.x;

  result[t].x   = t;
  result[t].y   = t+1;

  __syncthreads();
  rtype temp;
  for (unsigned offset = 1; offset < N; offset *= 2) {
    if (t >= offset) {
      temp = result[t-offset];
    }
    __syncthreads();
    if (t >= offset) {
      if ( !((temp.x < temp.y) &&
             (         temp.y == result[t].x) &&
             (                   result[t].x < result[t].y)) ) {
        *error = 1;
      }
      result[t].x = temp.x;
    }
    __syncthreads();
  }

  out[t] = result[t];
}

int main(int argc, char **argv) {
  // test data
  unsigned error;
  size_t ArraySize = N * sizeof(rtype);
  rtype *in  = (rtype *)malloc(ArraySize);
  rtype *out = (rtype *)malloc(ArraySize);
#ifdef _SYM
  klee_make_symbolic(in, ArraySize, "in");
  klee_make_symbolic(&error, sizeof(unsigned), "error");
  klee_assume(error == 0);
#else
  error = 0;
#endif

  // create arrays on device
  unsigned *d_error;
  rtype *d_in;
  rtype *d_out;
  ASSERT_NO_CUDA_ERROR(hipMalloc((void **)&d_error, sizeof(unsigned)));
  ASSERT_NO_CUDA_ERROR(hipMalloc((void **)&d_in, ArraySize));
  ASSERT_NO_CUDA_ERROR(hipMalloc((void **)&d_out, ArraySize));

  // memcpy into arrays
  ASSERT_NO_CUDA_ERROR(hipMemcpy(d_error, &error, sizeof(unsigned), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(hipMemcpy(d_in, in, ArraySize, hipMemcpyHostToDevice));

  // run the kernel
  ASSERT_NO_CUDA_ERROR(hipDeviceSynchronize());
#ifndef _SYM
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Pre-kernel error: %s.\n", hipGetErrorString(err));
    return 1;
  }
#endif
  koggestone<<<1,N>>>(d_in, d_out, d_error);
#ifndef _SYM
  ASSERT_NO_CUDA_ERROR(hipDeviceSynchronize());
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Post-kernel Error: %s\n", hipGetErrorString(err));
    return 1;
  }
#endif

  // memcpy back the result
  ASSERT_NO_CUDA_ERROR(hipMemcpy(&error, d_error, sizeof(unsigned), hipMemcpyDeviceToHost));
  ASSERT_NO_CUDA_ERROR(hipMemcpy(out, d_out, ArraySize, hipMemcpyDeviceToHost));

  // check monotonic specification
#ifdef _SYM
  unsigned i,j;
  klee_make_symbolic(&i, sizeof(unsigned), "i");
  klee_make_symbolic(&j, sizeof(unsigned), "j");
  klee_assume(i < N);
  klee_assume(j < N);
  klee_assume(i < j);
  if (error) {
    printf("TEST FAIL: ASSERTION FIRED\n");
    assert(false);
  }
  if (!( out[i].x == 0 && out[i].y == i+1 && out[i].y < out[j].y )) {
    printf("TEST FAIL: MONOTONIC SPECIFICATION\n");
    assert(false);
  }
#else
  // check full specification
  assert(error == 0);
  for (unsigned i=0; i<N; ++i) {
    printf("out[%d] = (%d,%d) (0,%d)\n", i, out[i].x, out[i].y, i+1);
    assert(out[i].x == 0);
    assert(out[i].y == i+1);
  }
#endif
  printf("TEST PASSED\n");

  // cleanup
  free(in);
  free(out);
  ASSERT_NO_CUDA_ERROR(hipFree(d_in));
  ASSERT_NO_CUDA_ERROR(hipFree(d_out));
  return 0;
}
